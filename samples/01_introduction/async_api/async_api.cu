
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

__global__ void increment(int* g_data, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
}

bool is_correct(int* data, int n, int x) {
    for (int i = 0; i < n; i++) {
        if (data[i] != x) {
            printf("Error: data[%d] != %d\n", i, x);
            return false;
        }
    }
    return true;
}

int main(int argc, char** argv) {
    int devId = 0;
    hipDeviceProp_t prop;
    hipGetDevice(&devId);

    hipGetDeviceProperties(&prop, devId);
    printf("Device: %s\n", prop.name);

    constexpr int n = 16 << 20;
    constexpr int nbytes = n * sizeof(int);
    int value = 26;

    int* a = 0;
    hipHostMalloc(&a, nbytes, hipHostMallocDefault);
    memset(a, 0, nbytes);

    int* d_a = 0;
    hipMalloc(&d_a, nbytes);
    hipMemset(d_a, 255, nbytes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n / threads.x, 1);
    hipDeviceSynchronize();

    float elapsed = 0.0f;
    hipProfilerStart();
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice);
    increment<<<(n + 255) / 256, 256>>>(d_a, value);
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipProfilerStop();

    uint32_t counter = 0;
    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }
    hipEventElapsedTime(&elapsed, start, stop);

    printf("Elapsed time: %f ms\n", elapsed);
    printf("Counter: %d\n", counter);

    if (is_correct(a, n, value)) {
        printf("Correct!\n");
    }

    hipFree(d_a);
    hipHostFree(a);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}