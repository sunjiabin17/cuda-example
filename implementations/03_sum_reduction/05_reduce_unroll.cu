#include <hip/hip_runtime.h>


#include <iostream>
#include <vector>
#include <algorithm>
#include <numeric>

using std::cout;
using std::endl;
using std::vector;
using std::accumulate;
using std::generate;

constexpr int THREADS = 256;
// note: 256
constexpr int SHARED_SIZE = 256;

__device__ void warpReduce(volatile int *shared, int tid) {
    shared[tid] += shared[tid + 32];
    shared[tid] += shared[tid + 16];
    shared[tid] += shared[tid + 8];
    shared[tid] += shared[tid + 4];
    shared[tid] += shared[tid + 2];
    shared[tid] += shared[tid + 1];
}

// 解决threads idle的问题
__global__ void reduce(int *input, int *output) {
    __shared__ int shared[SHARED_SIZE];
    
    int gid = blockIdx.x * (2*blockDim.x) + threadIdx.x;
    
    // thread 0: 计算input[0] + input[256] -> shared[0]
    // thread 1: 计算input[1] + input[257] -> shared[1]
    // ...
    // thread 255: 计算input[255] + input[511] -> shared[255]
    // 一个block处理512个元素
    shared[threadIdx.x] = input[gid] + input[gid + blockDim.x];
    __syncthreads();

    // note: i > 32
    // #pragma unroll
    for(unsigned int i=blockDim.x/2; i > 32; i >>= 1) {
        if (threadIdx.x < i) {
            shared[threadIdx.x] += shared[threadIdx.x + i];
        }
        __syncthreads();
    }
    // note: unroll warp 
    if (threadIdx.x < 32) {
        warpReduce(shared, threadIdx.x);
    }
    if (threadIdx.x == 0) {
        output[blockIdx.x] = shared[threadIdx.x];
    }
}

int main() {
    constexpr int N = 1 << 16;
    int size = N * sizeof(int);
    vector<int> h_input(N);
    vector<int> h_output(N);

    generate(h_input.begin(), h_input.end(), []() { return rand() % 100; });
    int h_sum = accumulate(h_input.begin(), h_input.end(), 0);

    int *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, h_input.data(), size, hipMemcpyHostToDevice);

    constexpr int BLOCKS = N / THREADS;

    // note: BLOCKS/2
    reduce<<<BLOCKS/2, THREADS>>>(d_input, d_output);

    reduce<<<1, BLOCKS>>>(d_output, d_output);

    hipMemcpy(h_output.data(), d_output, size, hipMemcpyDeviceToHost);


    cout << "Host sum: " << h_sum << endl;
    cout << "Device sum: " << h_output[0] << endl;

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}