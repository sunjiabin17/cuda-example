#include <hip/hip_runtime.h>


#include <iostream>
#include <vector>
#include <algorithm>


constexpr int M = 1 << 10;
constexpr int N = 1 << 9;
constexpr int K = 1 << 8;

// constexpr int THREADS = 32;

constexpr int tile_K = 32;
constexpr int tile_M = 16;  // 32
constexpr int tile_N = 64;  // 32

constexpr int BLOCK_M = M / tile_M;
constexpr int BLOCK_N = N / tile_N;


__global__ void gemm(int* A, int* B, int* C) {
    __shared__ int sA[tile_M][tile_K];
    __shared__ int sB[tile_K][tile_N];
    // 每个block负责C中一个维度bm*bn为的小矩阵块的计算
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // 方法2: 计算要copy到shared memory中的A和B的偏移位置
    A = &(A[tile_M*tile_K*(gridDim.x * by + bx)]);
    B = &(B[tile_K*tile_N*(gridDim.y * bx + by)]);

    int temp = 0.0f;
    for (int i = 0; i < K / tile_K; ++i) {
        // 方法1: 计算要copy到shared memory中的A和B的全局地址
        // // 每个线程负责A中一个维度bm*tk为的小矩阵块的计算
        // sA[ty][tx] = A[(by * tile_M + ty) * K + (i * tile_K + tx)];
        // // 每个线程负责B中一个维度tk*bn为的小矩阵块的计算
        // // sB[ty][tx] = B[(i * tile_K + ty) * N + (bx * tile_N + tx)];
        // sB[tx][ty] = B[(i * tile_K + ty) * N + (bx * tile_N + tx)];

        // 方法2: 计算要copy到shared memory中的A和B的偏移位置
        sA[ty][tx] = A[tx + ty * tile_K];
        sB[ty][tx] = B[tx + ty * tile_N];
        
        __syncthreads();
        
        for (int j = 0; j < tile_K; ++j) {
            // temp += sA[ty][j] * sB[j][tx];
            temp += sA[ty][j] * sB[tx][j];
        }
        
        __syncthreads();
    }

    C[(by * tile_M + ty) * N + (bx * tile_N + tx)] = temp;
}

int main() {
    int* A = new int[M * K];
    int* B = new int[K * N];
    int* C = new int[M * N];

    std::generate(A, A + M * K, []() {return (int)(rand() % 10); });
    std::generate(B, B + K * N, []() {return (int)(rand() % 10); });
    std::fill(C, C + M * N, 0.0f);

    // std::fill(A, A + M * K, 0.121f);
    // std::fill(B, B + K * N, 1.221f);
    // std::fill(C, C + M * N, 0.0f);

    int* dA, * dB, * dC;
    hipMalloc(&dA, M * K * sizeof(int));
    hipMalloc(&dB, K * N * sizeof(int));
    hipMalloc(&dC, M * N * sizeof(int));

    hipMemcpy(dA, A, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, K * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dC, C, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(tile_N, tile_M);
    dim3 grid(BLOCK_N, BLOCK_M);

    gemm<<<grid, block>>>(dA, dB, dC);

    hipMemcpy(C, dC, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // gemm on cpu
    bool flag = true;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            int c = 0.0f;
            for (int k = 0; k < K; ++k) {
                c += A[i * K + k] * B[k * N + j];
            }
            if (abs(C[i * N + j] - c) > 1) {
                flag = false;
                std::cout << i << " " << j << " " << C[i * N + j] << " " << c << std::endl;
                break;
            }
        }
        if (!flag) {
            break;
        }
    }
    if (flag) {
        std::cout << "correct" << std::endl;
    }
    else {
        std::cout << "wrong" << std::endl;
    }
    

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;



    return 0;
}
