#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

void init(int *a, int n) {
    for (int i = 0; i < n; i++)
        a[i] = rand() % 100;
}

void verify(int *c, int* ref_c, int n) {
    for (int i = 0; i < n; i++) {
        if (c[i] != ref_c[i]) {
            printf("Error at index %d: %d != %d", i, c[i], ref_c[i]);
            return;
        }
    }
    printf("Success!");
}


int main() {
    int device = -1;
    hipGetDevice(&device);

    constexpr int N = 1 << 16; // 65536 elements
    constexpr size_t bytes = sizeof(int) * N;

    int *a, *b, *c, *ref_c;
    // unified memory
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);
    
    ref_c = (int*)malloc(bytes);

    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, device);
    
    init(a, N);
    init(b, N);

    for (int i = 0; i < N; i++) {
        ref_c[i] = a[i] + b[i];
    }

    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, device);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, device);
    hipMemPrefetchAsync(a, bytes, device);
    hipMemPrefetchAsync(b, bytes, device);

    int NUM_THREADS = 1 << 10;
    
    // We need to launch at LEAST as many threads as we have elements
    // This equation pads an extra CTA to the grid if N cannot evenly be divided
    // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
    // int NUM_BLOCKS = (int)ceil(N / NUM_THREADS);

    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(a, b, c, N);
    hipDeviceSynchronize();

    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

    verify(c, ref_c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    free(ref_c);
    
    return 0;
}

// compile: nvcc -o prefetch thisfile.cu
// run: ./prefetch
// profile: nvprof ./prefetch
