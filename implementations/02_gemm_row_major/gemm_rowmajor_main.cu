#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <vector>
#include <algorithm>
#include <random>

#include "gemm1_naive.cuh"

using namespace std;

#define M 1024
#define N 1024
#define K 1024

int main(int argc, char **argv) {
    float* A = new float[M * K];
    float* B = new float[K * N];
    float* C = new float[M * N];
    float* C1 = new float[M * N];

    // set random seed
    srand((unsigned)time(NULL));

    std::generate(A, A + M * K, []() {return (float)(rand() % 10); });
    std::generate(B, B + K * N, []() {return (float)(rand() % 10); });
    std::fill(C, C + M * N, 0.0f);

    float* dA, * dB, * dC;
    hipMalloc(&dA, M * K * sizeof(float));
    hipMalloc(&dB, K * N * sizeof(float));
    hipMalloc(&dC, M * N * sizeof(float));

    hipMemcpy(dA, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dC, C, M * N * sizeof(float), hipMemcpyHostToDevice);

    // matrix multiplication using cublas (M, K) * (K, N) = (M, N)
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f;
    float beta = 0.0f;
    
    int lda = K;
    int ldb = N;
    int ldc = N;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dB, ldb, dA, lda, &beta, dC, ldc);
    hipMemcpy(C, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);

    test_sgemm1(M, N, K, &alpha, dA, lda, dB, ldb, &beta, dC, ldc);
    hipMemcpy(C1, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);    
    

    // compare
    bool flag = true;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            if (abs(C[i * N + j] - C1[i * N + j]) > 1e-5) {
                std::cout << i << " " << j << " " << C[i * N + j] << " " << C1[i * N + j] << std::endl;
                flag = false;
                break;
            }
        }
        if (!flag) {
            break;
        }
    }
    if (flag) {
        std::cout << "correct" << std::endl;
    } else {
        std::cout << "wrong" << std::endl;
    }

    if (false) {
        // print A
        std::cout << "A: " << std::endl;
        for (int i = 0; i < M; ++i) {
            for (int j = 0; j < K; ++j) {
                std::cout << A[i * K + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        // print B
        std::cout << "B: " << std::endl;
        for (int i = 0; i < K; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << B[i * N + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        // print C
        std::cout << "C: " << std::endl;
        for (int i = 0; i < M; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << C[i * N + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        // print C1
        std::cout << "C1: " << std::endl;
        for (int i = 0; i < M; ++i) {
            for (int j = 0; j < N; ++j) {
                std::cout << C1[i * N + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}