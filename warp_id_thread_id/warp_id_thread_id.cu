#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void print_details_of_warps() {
    int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    int warp_id = threadIdx.x / 32;
    
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;

    printf("gid: %d, warp_id: %d, gbid: %d\n", gid, warp_id, gbid);
}

int main() {
    dim3 block_size(42);
    dim3 grid_size(2,2);

    print_details_of_warps<<<grid_size, block_size>>>();
    
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}