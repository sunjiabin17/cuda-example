#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "common.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void scan_efficient_1G(int *input, int *auxiliry_array, int input_size);
__global__ void scan_summation(int *input, int *auxiliry_array, int input_size);

// reduction neighbored pairs kernel
__global__ void redunction_neighbored_pairs(int *input,
                                            int *temp, int size)
{
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    if (gid > size)
        return;

    for (int offset = 1; offset <= blockDim.x / 2; offset *= 2)
    {
        if (tid % (2 * offset) == 0)
        {
            input[gid] += input[gid + offset];
        }

        __syncthreads();
    }

    if (tid == 0)
    {
        temp[blockIdx.x] = input[gid];
    }
}

// warp divergence improved
__global__ void reduction_neighbored_pairs_improved(
    int *int_array, int *temp_array, int size)
{
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // local data block pointer
    int *i_data = int_array + blockDim.x * blockIdx.x;

    if (gid > size)
        return;

    for (int offset = 1; offset <= blockDim.x / 2; offset *= 2)
    {
        int index = 2 * offset * tid;

        if (index < blockDim.x)
        {
            i_data[index] += i_data[index + offset];
        }

        __syncthreads();
    }

    if (tid == 0)
    {
        temp_array[blockIdx.x] = int_array[gid];
    }
}

//kernel interleaved pairs
__global__ void reduction_interleaved_pairs(int * int_array,
    int * temp_array, int size)
{
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    if (gid > size)
        return;

    for (int offset = blockDim.x/ 2; offset > 0; offset = offset/2)
    {
        if (tid < offset)
        {
            int_array[gid] += int_array[gid + offset];
        }

        __syncthreads();
    }

    if (tid == 0)
    {
        temp_array[blockIdx.x] = int_array[gid];
    }
}

// loop unrolling
__global__ void reduction_kernel_warp_unrolling(int * int_array,
    int * temp_array, int size)
{
    int tid = threadIdx.x;

    //element index for this thread
    int index = blockDim.x * blockIdx.x  + threadIdx.x;

    //local data pointer
    int * i_data = int_array + blockDim.x * blockIdx.x ;

    for (int offset = blockDim.x/2; offset >= 64; offset = offset/2)
    {
        if (tid < offset)
        {
            i_data[tid] += i_data[tid + offset];
        }
        __syncthreads();
    }
		// 展开 unrolling last 5 iteration
    if (tid < 32)
    {
        volatile int * vsmem = i_data;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0)
    {
        temp_array[blockIdx.x] = i_data[0];
    }
}

// warp unrolling
__global__ void reduction_kernel_warp_unrolling(int * int_array,
    int * temp_array, int size)
{
    int tid = threadIdx.x;

    //element index for this thread
    int index = blockDim.x * blockIdx.x  + threadIdx.x;

    //local data pointer
    int * i_data = int_array + blockDim.x * blockIdx.x ;

    for (int offset = blockDim.x/2; offset >= 64; offset = offset/2)
    {
        if (tid < offset)
        {
            i_data[tid] += i_data[tid + offset];
        }
        __syncthreads();
    }
		// 展开 unrolling last 5 iteration
    if (tid < 32)
    {
        volatile int * vsmem = i_data;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0)
    {
        temp_array[blockIdx.x] = i_data[0];
    }
}

int main(int argc, char **argv)
{
    printf("Running neighbored pairs reduction kernel \n");

    int size = 1 << 27; // 128 Mb of data
    int byte_size = size * sizeof(int);
    int block_size = 128;

    int *h_input, *h_ref;
    h_input = (int *)malloc(byte_size);

    initialize(h_input, size, INIT_RANDOM);

    // get the reduction result from cpu
    int cpu_result = reduction_cpu(h_input, size);

    dim3 block(block_size);
    dim3 grid(size / block.x);

    printf("Kernel launch parameters | grid.x : %d, block.x : %d \n",
           grid.x, block.x);

    int temp_array_byte_size = sizeof(int) * grid.x;
    h_ref = (int *)malloc(temp_array_byte_size);

    int *d_input, *d_temp;

    gpuErrchk(hipMalloc((void **)&d_input, byte_size));
    gpuErrchk(hipMalloc((void **)&d_temp, temp_array_byte_size));

    gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
    gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
                         hipMemcpyHostToDevice));

    redunction_neighbored_pairs<<<grid, block>>>(d_input, d_temp, size);

    gpuErrchk(hipDeviceSynchronize());

    hipMemcpy(h_ref, d_temp, temp_array_byte_size,
               hipMemcpyDeviceToHost);

    int gpu_result = 0;

    for (int i = 0; i < grid.x; i++)
    {
        gpu_result += h_ref[i];
    }

    // validity check
    compare_results(gpu_result, cpu_result);

    gpuErrchk(hipFree(d_temp));
    gpuErrchk(hipFree(d_input));

    free(h_ref);
    free(h_input);

    gpuErrchk(hipDeviceReset());
    return 0;
}