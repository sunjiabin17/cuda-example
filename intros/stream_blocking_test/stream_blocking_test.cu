#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


__global__ void blocking_nonblocking_test1()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		for (size_t i = 0; i < 10000; i++)
		{
			printf("kernel 1 \n");
		}
	}
}

int main(int argc, char **argv)
{
	int size = 1 << 15;

	hipStream_t stm1, stm2, stm3;
	hipStreamCreateWithFlags(&stm1, hipStreamNonBlocking);
	hipStreamCreate(&stm2);
	hipStreamCreateWithFlags(&stm3, hipStreamNonBlocking);

	dim3 block(128);
	dim3 grid(size / block.x);

	blocking_nonblocking_test1<<<grid, block, 0, stm1>>>();
	blocking_nonblocking_test1<<<grid, block>>>();
	blocking_nonblocking_test1<<<grid, block, 0, stm3>>>();

	hipStreamDestroy(stm1);
	hipStreamDestroy(stm2);
	hipStreamDestroy(stm3);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}